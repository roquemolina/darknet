#include "hip/hip_runtime.h"
#include "darknet_internal.hpp"
#include "gemm.hpp"
#include "col2im.hpp"
#include "im2col.hpp"


namespace
{
	static auto & cfg_and_state = Darknet::CfgAndState::get();
}


__global__ void binarize_kernel(float *x, int n, float *binary)
{
	int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
	if (i >= n) return;
	binary[i] = (x[i] >= 0) ? 1 : -1;
}

void binarize_gpu(float *x, int n, float *binary)
{
	TAT(TATPARMS);

	binarize_kernel<<<cuda_gridsize(n), BLOCK, 0, get_cuda_stream() >>>(x, n, binary);
	CHECK_CUDA(hipPeekAtLastError());
}

__global__ void binarize_input_kernel(float *input, int n, int size, float *binary)
{
	int s = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
	if (s >= size) return;
	int i = 0;
	float mean = 0;
	for(i = 0; i < n; ++i){
		mean += fabs(input[i*size + s]);
	}
	mean = mean / n;
	for(i = 0; i < n; ++i){
		binary[i*size + s] = (input[i*size + s] > 0) ? mean : -mean;
	}
}

void binarize_input_gpu(float *input, int n, int size, float *binary)
{
	TAT(TATPARMS);

	binarize_input_kernel<<<cuda_gridsize(size), BLOCK, 0, get_cuda_stream() >>>(input, n, size, binary);
	CHECK_CUDA(hipPeekAtLastError());
}

__global__ void binarize_weights_kernel(float *weights, int n, int size, float *binary)
{
	int f = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
	if (f >= n) return;
	int i = 0;
	float mean = 0;
	for (i = 0; i < size; ++i)
	{
		mean += fabs(weights[f*size + i]);
	}
	mean = mean / size;
	for (i = 0; i < size; ++i)
	{
		binary[f*size + i] = (weights[f*size + i] > 0) ? mean : -mean;
	}
}

void binarize_weights_gpu(float *weights, int n, int size, float *binary)
{
	TAT(TATPARMS);

	binarize_weights_kernel <<<cuda_gridsize(n), BLOCK, 0, get_cuda_stream() >>>(weights, n, size, binary);
	CHECK_CUDA(hipPeekAtLastError());
}


__global__ void set_zero_kernel(float *src, int size)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < size) src[i] = 0;
}

__inline__ __device__ float warpAllReduceSum(float val)
{
	for (int mask = WARP_SIZE / 2; mask > 0; mask /= 2)
#if CUDART_VERSION >= 9000
		val += __shfl_xor_sync(0xffffffff, val, mask);
#else
		val += __shfl_xor(val, mask);
#endif
	return val;
}

// only if (size % 32 == 0)
__global__ void reduce_kernel(float *weights, int n, int size, float *mean_arr_gpu)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int f = i / size;
	if (f >= n) return;
	float warp_mean = warpAllReduceSum(fabs(weights[i]));
	if (i % 32 == 0)
	{
		atomicAdd(&mean_arr_gpu[f], warp_mean / size);
	}
}

__global__ void binarize_weights_mean_kernel(float *weights, int n, int size, float *binary, float *mean_arr_gpu)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int f = i / size;
	if (f >= n) return;
	float mean = mean_arr_gpu[f];
	binary[i] = (weights[i] > 0) ? mean : -mean;
}

void fast_binarize_weights_gpu(float *weights, int n, int size, float *binary, float *mean_arr_gpu)
{
	TAT(TATPARMS);

	if (size % 32 == 0) {
		size_t gridsize = n * size;
		const int num_blocks = get_number_of_blocks(gridsize, BLOCK);// gridsize / BLOCK + 1;

		set_zero_kernel <<<(n/BLOCK + 1), BLOCK, 0, get_cuda_stream() >>> (mean_arr_gpu, n);
		reduce_kernel <<<num_blocks, BLOCK, 0, get_cuda_stream() >>> (weights, n, size, mean_arr_gpu);
		binarize_weights_mean_kernel <<<num_blocks, BLOCK, 0, get_cuda_stream() >>> (weights, n, size, binary, mean_arr_gpu);
		CHECK_CUDA(hipPeekAtLastError());
	}
	else {
		binarize_weights_gpu(weights, n, size, binary);
	}
}


__global__ void cuda_f32_to_f16(float* input_f32, size_t size, half *output_f16)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < size) output_f16[idx] = __float2half(input_f32[idx]);
}

void cuda_convert_f32_to_f16(float* input_f32, size_t size, float *output_f16)
{
	TAT(TATPARMS);

	cuda_f32_to_f16 <<< get_number_of_blocks(size, BLOCK), BLOCK, 0, get_cuda_stream() >>> (input_f32, size, (half *)output_f16);
	CHECK_CUDA(hipPeekAtLastError());
}

__global__ void cuda_f16_to_f32(half* input_f16, size_t size, float *output_f32)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < size) output_f32[idx] = __half2float(input_f16[idx]);
}

void cuda_convert_f16_to_f32(float* input_f16, size_t size, float *output_f32)
{
	TAT(TATPARMS);

	cuda_f16_to_f32 <<< get_number_of_blocks(size, BLOCK), BLOCK, 0, get_cuda_stream() >>> ((half *)input_f16, size, output_f32);
	CHECK_CUDA(hipPeekAtLastError());
}

half *cuda_make_f16_from_f32_array(float *src, size_t n)
{
	TAT(TATPARMS);

	half *dst16;
	size_t size = sizeof(half)*n;
	CHECK_CUDA(hipMalloc((void **)&dst16, size));
	if (src) {
		assert(n > 0);
		cuda_convert_f32_to_f16(src, n, (float *)dst16);
	}
	if (!dst16)
	{
		darknet_fatal_error(DARKNET_LOC, "CUDA malloc failed (n=%d)", n);
	}
	return dst16;
}

void forward_convolutional_layer_gpu(Darknet::Layer & l, Darknet::NetworkState state)
{
	TAT(TATPARMS);

	if (l.train == 0) state.train = 0;

	if (l.stream >= 0) {
		switch_stream(l.stream);
	}

	if (l.wait_stream_id >= 0) {
		wait_stream(l.wait_stream_id);
	}

	//fill_ongpu(l.outputs*l.batch, 0, l.output_gpu, 1);
	if (l.binary)
	{
		binarize_weights_gpu(l.weights_gpu, l.n, (l.c / l.groups)*l.size*l.size, l.binary_weights_gpu);
		swap_binary(&l);
	}

	if (l.xnor)
	{
		if (!l.align_bit_weights_gpu || state.train)
		{
			fast_binarize_weights_gpu(l.weights_gpu, l.n, (l.c / l.groups)*l.size*l.size, l.binary_weights_gpu, l.mean_arr_gpu);
		}

		if (l.align_bit_weights_gpu && !state.train && l.c >= 32 && l.stride_x == l.stride_y)
		{
			int m = l.n / l.groups;
			int k = l.size*l.size*l.c / l.groups;
			int n = l.out_w*l.out_h;

			const int ldb_align = l.lda_align;
			const size_t new_ldb = k + (ldb_align - k%ldb_align); // (k / 8 + 1) * 8;

			if (l.c % 32 == 0)
			{
				const int new_c = l.c / 32;

				repack_input_gpu_bin(state.input, (uint32_t *)l.align_workspace_gpu, l.w, l.h, l.c);

				im2col_ongpu(l.align_workspace_gpu, new_c, l.h, l.w, l.size, l.stride, l.pad, state.workspace);

				int new_k = l.size*l.size*l.c / 32;

				transpose_uint32_gpu((uint32_t *)state.workspace, (uint32_t *)l.transposed_align_workspace_gpu, new_k, n, n, new_ldb);
				gemm_nn_custom_bin_mean_transposed_gpu(m, n, k,
					(unsigned char *)l.align_bit_weights_gpu, new_ldb, (unsigned char *)l.transposed_align_workspace_gpu,
					new_ldb, l.output_gpu, n, l.mean_arr_gpu, l.biases_gpu, l.activation == LEAKY,
					l.bin_conv_shortcut_in_gpu, l.bin_conv_shortcut_out_gpu);
			}
			else
			{
				int i = 0;
				{
					im2col_align_ongpu(state.input + i*l.c*l.h*l.w, l.c, l.h, l.w, l.size, l.stride, l.pad, l.align_workspace_gpu, l.bit_align);

					// should be optimized
					float_to_bit_gpu(l.align_workspace_gpu, (unsigned char *)state.workspace, l.align_workspace_size);
				}
				transpose_bin_gpu((unsigned char *)state.workspace, (unsigned char *)l.transposed_align_workspace_gpu, k, n, l.bit_align, new_ldb, 8);

				gemm_nn_custom_bin_mean_transposed_gpu(m, n, k,
						(unsigned char *)l.align_bit_weights_gpu, new_ldb, (unsigned char *)l.transposed_align_workspace_gpu,
						new_ldb, l.output_gpu, n, l.mean_arr_gpu, l.biases_gpu, l.activation == LEAKY,
						l.bin_conv_shortcut_in_gpu, l.bin_conv_shortcut_out_gpu);
			}

			if (l.activation == SWISH) activate_array_swish_ongpu(l.output_gpu, l.outputs*l.batch, l.activation_input_gpu, l.output_gpu);
			else if (l.activation == MISH) activate_array_mish_ongpu(l.output_gpu, l.outputs*l.batch, l.activation_input_gpu, l.output_gpu);
			else if (l.activation == HARD_MISH) activate_array_hard_mish_ongpu(l.output_gpu, l.outputs*l.batch, l.activation_input_gpu, l.output_gpu);
			else if (l.activation == NORM_CHAN) activate_array_normalize_channels_ongpu(l.output_gpu, l.outputs*l.batch, l.batch, l.out_c, l.out_w*l.out_h, l.output_gpu);
			else if (l.activation == NORM_CHAN_SOFTMAX) activate_array_normalize_channels_softmax_ongpu(l.output_gpu, l.outputs*l.batch, l.batch, l.out_c, l.out_w*l.out_h, l.output_gpu, 0);
			else if (l.activation == NORM_CHAN_SOFTMAX_MAXVAL) activate_array_normalize_channels_softmax_ongpu(l.output_gpu, l.outputs*l.batch, l.batch, l.out_c, l.out_w*l.out_h, l.output_gpu, 1);
			else if (l.activation != LINEAR && l.activation != LEAKY) activate_array_ongpu(l.output_gpu, l.outputs*l.batch, l.activation);
			return;
		}
	}

	if (l.xnor)
	{
		swap_binary(&l);
		binarize_gpu(state.input, l.c*l.h*l.w*l.batch, l.binary_input_gpu);
		state.input = l.binary_input_gpu;
	}

	//fill_ongpu(l.outputs*l.batch, 0, l.output_gpu, 1);

#ifdef CUDNN
	//float one = 1;    // alpha[0], beta[0] is float for HALF and FLOAT
	float alpha = 1, beta = 0;

//#ifdef CUDNN_HALF
	//if (state.use_mixed_precision) {
	int iteration_num = get_current_iteration(state.net); // (*state.net.seen) / (state.net.batch*state.net.subdivisions);
	if (state.index != 0 && state.net.cudnn_half && !l.xnor && (!state.train || (iteration_num > 3 * state.net.burn_in) && state.net.loss_scale != 1) &&
		(l.c / l.groups) % 8 == 0 && l.n % 8 == 0 && l.groups <= 1 && l.size > 1)
	{
		// Note: For improved performance it is advised to use beta[0] = 0.0.
		// For Tensor Core: hipdnnSetConvolutionMathType() where hipdnnMathType_t mathType = HIPDNN_TENSOR_OP_MATH;
		// 1. or HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_PRECOMP_GEMM and use HIPDNN_DATA_HALF
		// 2. or HIPDNN_CONVOLUTION_FWD_ALGO_WINOGRAD_NONFUSED
		// More: http://docs.nvidia.com/deeplearning/sdk/cudnn-developer-guide/index.html#tensor_ops

		const size_t input16_size = l.batch*l.c*l.w*l.h;
		const size_t output16_size = l.batch*l.out_c*l.out_h*l.out_w;

		if (*state.net.max_input16_size < input16_size)
		{
			*state.net.max_input16_size = input16_size;
			if (*state.net.input16_gpu) cuda_free(*state.net.input16_gpu);
			assert(*state.net.max_input16_size > 0);
			*state.net.input16_gpu = (float *)cuda_make_f16_from_f32_array(NULL, *state.net.max_input16_size);
		}
		float *input16 = *state.net.input16_gpu;

		if (*state.net.max_output16_size < output16_size) {
			*state.net.max_output16_size = output16_size;
			if (*state.net.output16_gpu) cuda_free(*state.net.output16_gpu);
			assert(*state.net.max_output16_size > 0);
			*state.net.output16_gpu = (float *)cuda_make_f16_from_f32_array(NULL, *state.net.max_output16_size);
		}
		float *output16 = *state.net.output16_gpu;

		assert(input16_size > 0);
		cuda_convert_f32_to_f16(state.input, input16_size, input16);

		CHECK_CUDNN(hipdnnConvolutionForward(cudnn_handle(),
			&alpha,
			l.srcTensorDesc16,
			input16,
			l.weightDesc16,
			l.weights_gpu16,
			l.convDesc,
			l.fw_algo16,
			state.workspace,
			l.workspace_size,
			&beta,
			l.dstTensorDesc16,
			output16));


		if (l.batch_normalize)
		{
			if (state.train && !state.net.adversarial) // Training
			{
				simple_copy_ongpu(l.outputs*l.batch / 2, output16, l.x_gpu);
				float one = 1.0f;
				float zero = 0.0f;
				// Batch-normalization can still take FP16 inputs and outputs, saving half the bandwidth
				// compared to FP32, it's just that the statistics and value adjustment should be done in FP32.
				CHECK_CUDNN(hipdnnBatchNormalizationForwardTraining(cudnn_handle(),
					HIPDNN_BATCHNORM_SPATIAL,
					&one,
					&zero,
					l.normDstTensorDescF16,
					l.x_gpu,            // input
					l.normDstTensorDescF16,
					output16,            // output
					l.normTensorDesc,
					l.scales_gpu,       // input
					l.biases_gpu,       // input
					.01,
					l.rolling_mean_gpu,        // input/output (should be FP32)
					l.rolling_variance_gpu,    // input/output (should be FP32)
					.00001,
					l.mean_gpu,            // output (should be FP32) - optional cache to speedup hipdnnBatchNormalizationBackward()
					l.variance_gpu));    // output (should be FP32) - optional cache to speedup hipdnnBatchNormalizationBackward()

				cuda_convert_f16_to_f32(output16, output16_size, l.output_gpu);
				//forward_batchnorm_layer_gpu(l, state);
			}
			else // Detection
			{
				cuda_convert_f16_to_f32(output16, output16_size, l.output_gpu);
				normalize_gpu(l.output_gpu, l.rolling_mean_gpu, l.rolling_variance_gpu, l.batch, l.out_c, l.out_h*l.out_w);
				scale_bias_gpu(l.output_gpu, l.scales_gpu, l.batch, l.out_c, l.out_h*l.out_w);
				add_bias_gpu(l.output_gpu, l.biases_gpu, l.batch, l.out_c, l.out_w*l.out_h);
			}
		}
		else // BIAS only
		{
			cuda_convert_f16_to_f32(output16, output16_size, l.output_gpu);
			add_bias_gpu(l.output_gpu, l.biases_gpu, l.batch, l.n, l.out_w*l.out_h);
		}
	}
	else
	{
		CHECK_CUDNN(hipdnnConvolutionForward(cudnn_handle(),
			&alpha, //&one,
			l.srcTensorDesc,
			state.input,
			l.weightDesc,
			l.weights_gpu,
			l.convDesc,
			l.fw_algo,
			state.workspace,
			l.workspace_size,
			&beta,  //&one,
			l.dstTensorDesc,
			l.output_gpu));

		//hipDeviceSynchronize();
		if (l.batch_normalize) {
			forward_batchnorm_layer_gpu(l, state);
		}
		else {
			add_bias_gpu(l.output_gpu, l.biases_gpu, l.batch, l.n, l.out_w*l.out_h);
		}
	//#endif    // CUDNN_HALF
	}


#else
	fill_ongpu(l.outputs*l.batch, 0, l.output_gpu, 1);

	int i, j;
	int m = l.n / l.groups;
	int k = l.size*l.size*l.c / l.groups;
	int n = l.out_w*l.out_h;
	for(i = 0; i < l.batch; ++i){
		for (j = 0; j < l.groups; ++j) {
			//float *im = state.input + i*l.c*l.h*l.w;
			float *im = state.input + (i*l.groups + j)*l.c / l.groups*l.h*l.w;
			float *a = l.weights_gpu + j*l.nweights / l.groups;
			float *b = state.workspace;
			float *c = l.output_gpu + (i*l.groups + j)*n*m;
			if (l.size == 1 && l.stride == 1 && l.dilation == 1) {
				b = im;
			}
			else {
				//im2col_ongpu(im, l.c / l.groups, l.h, l.w, l.size, l.stride, l.pad, state.workspace);

				im2col_gpu_ext(im,          // input
					l.c / l.groups,         // input channels
					l.h, l.w,               // input size (h, w)
					l.size, l.size,         // kernel size (h, w)
					l.pad * l.dilation, l.pad * l.dilation,   // padding (h, w)
					l.stride_y, l.stride_x,     // stride (h, w)
					l.dilation, l.dilation, // dilation (h, w)
					state.workspace);       // output

			}
			//gemm_ongpu(0, 0, m, n, k, 1., a, k, b, n, 1., c + i*m*n, n);
			gemm_ongpu(0, 0, m, n, k, 1, a, k, b, n, 1, c, n);
		}
	}

	if (l.batch_normalize) {
		forward_batchnorm_layer_gpu(l, state);
	}
	else {
		add_bias_gpu(l.output_gpu, l.biases_gpu, l.batch, l.n, l.out_w*l.out_h);
	}
#endif

//#ifndef CUDNN_HALF
//#endif // no CUDNN_HALF

	if (l.activation == SWISH) activate_array_swish_ongpu(l.output_gpu, l.outputs*l.batch, l.activation_input_gpu, l.output_gpu);
	else if (l.activation == MISH) activate_array_mish_ongpu(l.output_gpu, l.outputs*l.batch, l.activation_input_gpu, l.output_gpu);
	else if (l.activation == HARD_MISH) activate_array_hard_mish_ongpu(l.output_gpu, l.outputs*l.batch, l.activation_input_gpu, l.output_gpu);
	else if (l.activation == NORM_CHAN) activate_array_normalize_channels_ongpu(l.output_gpu, l.outputs*l.batch, l.batch, l.out_c, l.out_w*l.out_h, l.output_gpu);
	else if (l.activation == NORM_CHAN_SOFTMAX) activate_array_normalize_channels_softmax_ongpu(l.output_gpu, l.outputs*l.batch, l.batch, l.out_c, l.out_w*l.out_h, l.output_gpu, 0);
	else if (l.activation == NORM_CHAN_SOFTMAX_MAXVAL) activate_array_normalize_channels_softmax_ongpu(l.output_gpu, l.outputs*l.batch, l.batch, l.out_c, l.out_w*l.out_h, l.output_gpu, 1);
	else if (l.activation != LINEAR) activate_array_ongpu(l.output_gpu, l.outputs*l.batch, l.activation);
	//if(l.dot > 0) dot_error_gpu(l);
	if(l.binary || l.xnor) swap_binary(&l);
	//hipDeviceSynchronize();    // for correct profiling of performance

	if (state.net.try_fix_nan) {
		fix_nan_and_inf(l.output_gpu, l.outputs*l.batch);
	}

	if(l.assisted_excitation && state.train) assisted_excitation_forward_gpu(l, state);

	if (l.antialiasing) {
		Darknet::NetworkState s = { 0 };
		s.train = state.train;
		s.workspace = state.workspace;
		s.net = state.net;
		if (!state.train) s.index = state.index;  // don't use TC for training (especially without cuda_convert_f32_to_f16() )
		s.input = l.output_gpu;
		forward_convolutional_layer_gpu(*(l.input_layer), s);
		simple_copy_ongpu(l.outputs*l.batch, l.output_gpu, l.input_antialiasing_gpu);
		simple_copy_ongpu(l.input_layer->outputs*l.input_layer->batch, l.input_layer->output_gpu, l.output_gpu);
	}

	if (l.coordconv) {
		coord_conv_gpu(l.output_gpu, l.outputs*l.batch, l.out_w, l.out_h, l.out_c, l.batch, 0);
	}
}

void backward_convolutional_layer_gpu(Darknet::Layer & l, Darknet::NetworkState state)
{
	TAT(TATPARMS);

	if (l.coordconv) {
		coord_conv_gpu(l.delta_gpu, l.outputs*l.batch, l.out_w, l.out_h, l.out_c, l.batch, 1);
	}

	if (l.antialiasing) {
		Darknet::NetworkState s = { 0 };
		s.train = state.train;
		s.workspace = state.workspace;
		s.net = state.net;
		s.delta = l.delta_gpu;  // s.delta will be returned to l.delta_gpu
		s.input = l.input_antialiasing_gpu;
		//if (!state.train) s.index = state.index;  // don't use TC for training (especially without cuda_convert_f32_to_f16() )
		simple_copy_ongpu(l.input_layer->outputs*l.input_layer->batch, l.delta_gpu, l.input_layer->delta_gpu);
		backward_convolutional_layer_gpu(*(l.input_layer), s);

		simple_copy_ongpu(l.outputs*l.batch, l.input_antialiasing_gpu, l.output_gpu);
	}

	if(state.net.try_fix_nan) constrain_ongpu(l.outputs*l.batch, 1, l.delta_gpu, 1);

	if (l.activation == SWISH) gradient_array_swish_ongpu(l.output_gpu, l.outputs*l.batch, l.activation_input_gpu, l.delta_gpu);
	else if (l.activation == MISH) gradient_array_mish_ongpu(l.outputs*l.batch, l.activation_input_gpu, l.delta_gpu);
	else if (l.activation == HARD_MISH) gradient_array_hard_mish_ongpu(l.outputs*l.batch, l.activation_input_gpu, l.delta_gpu);
	else if (l.activation == NORM_CHAN_SOFTMAX || l.activation == NORM_CHAN_SOFTMAX_MAXVAL) gradient_array_normalize_channels_softmax_ongpu(l.output_gpu, l.outputs*l.batch, l.batch, l.out_c, l.out_w*l.out_h, l.delta_gpu);
	else if (l.activation == NORM_CHAN) gradient_array_normalize_channels_ongpu(l.output_gpu, l.outputs*l.batch, l.batch, l.out_c, l.out_w*l.out_h, l.delta_gpu);
	else gradient_array_ongpu(l.output_gpu, l.outputs*l.batch, l.activation, l.delta_gpu);

	if (!l.batch_normalize)
		backward_bias_gpu(l.bias_updates_gpu, l.delta_gpu, l.batch, l.n, l.out_w*l.out_h);

//#ifndef CUDNN_HALF
	//if(l.batch_normalize){
	//    backward_batchnorm_layer_gpu(l, state);
	//} else {
	//    //backward_bias_gpu(l.bias_updates_gpu, l.delta_gpu, l.batch, l.n, l.out_w*l.out_h);
	//}
//#endif // no CUDNN_HALF
	float *original_input = state.input;

	if(l.xnor) state.input = l.binary_input_gpu;
#ifdef CUDNN
	float alpha = 1.0f;
	float beta = 0.0f;

//#ifdef CUDNN_HALF
	int iteration_num = get_current_iteration(state.net); //(*state.net.seen) / (state.net.batch*state.net.subdivisions);
	if (state.index != 0 && state.net.cudnn_half && !l.xnor && (!state.train || (iteration_num > 3 * state.net.burn_in) && state.net.loss_scale != 1) &&
		(l.c / l.groups) % 8 == 0 && l.n % 8 == 0  && l.groups <= 1 && l.size > 1)
	{
		const size_t input16_size = l.batch*l.c*l.w*l.h;
		const size_t delta16_size = l.batch*l.n*l.out_w*l.out_h;

		if (*state.net.max_input16_size < input16_size) {
			*state.net.max_input16_size = input16_size;
			if (*state.net.input16_gpu) cuda_free(*state.net.input16_gpu);
			assert(*state.net.max_input16_size > 0);
			*state.net.input16_gpu = (float *)cuda_make_f16_from_f32_array(NULL, *state.net.max_input16_size);
		}
		float *input16 = *state.net.input16_gpu;

		if (*state.net.max_output16_size < delta16_size) {
			*state.net.max_output16_size = delta16_size;
			if (*state.net.output16_gpu) cuda_free(*state.net.output16_gpu);
			assert(*state.net.max_output16_size > 0);
			*state.net.output16_gpu = (float *)cuda_make_f16_from_f32_array(NULL, *state.net.max_output16_size);
		}
		float *delta16 = *state.net.output16_gpu;

		assert(input16_size > 0);
		assert(delta16_size > 0);
		cuda_convert_f32_to_f16(state.input, input16_size, input16);
		cuda_convert_f32_to_f16(l.delta_gpu, delta16_size, delta16);

		if (l.batch_normalize)
		{
			float one = 1.0f;
			float zero = 0.0f;
			CHECK_CUDNN(hipdnnBatchNormalizationBackward(cudnn_handle(),
				HIPDNN_BATCHNORM_SPATIAL,
				&one,
				&zero,
				&one,
				&one,
				l.normDstTensorDescF16,
				l.x_gpu,                // input (input in BN-forward-inference)
				l.normDstTensorDescF16,
				delta16,                // input
				l.normDstTensorDescF16,
				l.output_gpu, //l.x_norm_gpu,            // output (new delta)
				l.normTensorDesc,
				l.scales_gpu,            // input (should be FP32)
				l.scale_updates_gpu,    // output (should be FP32)
				l.bias_updates_gpu,        // output (should be FP32)
				.00001,
				l.mean_gpu,                // input (should be FP32)
				l.variance_gpu));        // input (should be FP32)

			simple_copy_ongpu(l.outputs*l.batch / 2, l.output_gpu, delta16);
		}

		// convert input: state.input (x), l.delta_gpu (y) from fp32 to fp16
		// get output: l.weight_updates_gpu (dw) and convert it to fp32 (ONLY if it is fp16)

		// calculate conv weight updates
		// Already: l.weight_updates_gpu = (l.weight_updates_gpu - l.weight*decay*batch*subdivision)*momentum
		//   so we should copy f32 to f16, or compute: f16=(w_up - w*d*b*s)*m
		assert((l.nweights) > 0);
		cuda_convert_f32_to_f16(l.weight_updates_gpu, l.nweights, l.weight_updates_gpu16);

		float one = 1.0f;
		if (!state.net.adversarial && !l.train_only_bn) {
			CHECK_CUDNN(hipdnnConvolutionBackwardFilter(cudnn_handle(),
				&one,
				l.srcTensorDesc16,
				input16, //state.input,
				l.ddstTensorDesc16,
				delta16, //l.delta_gpu,
				l.convDesc,
				l.bf_algo16,
				state.workspace,
				l.workspace_size,
				&one,
				l.dweightDesc16,
				l.weight_updates_gpu16));    // l.weight_updates_gpu);

			cuda_convert_f16_to_f32(l.weight_updates_gpu16, l.nweights, l.weight_updates_gpu);
		}

		if (state.delta) {
			if (l.binary || l.xnor) swap_binary(&l);

			// http://docs.nvidia.com/deeplearning/sdk/cudnn-developer-guide/index.html#hipdnnConvolutionBackwardData
			// calculate delta for the next layer
			// convert input: l.weights_gpu (w), l.delta_gpu (dy) from fp32 to fp16
			// get output: state.delta (dx) and convert it to fp32 (ONLY if it is fp16)
			CHECK_CUDNN(hipdnnConvolutionBackwardData(cudnn_handle(),
				&alpha,
				l.weightDesc16,
				l.weights_gpu16, //l.weights_gpu,
				l.ddstTensorDesc16,
				delta16, //l.delta_gpu,
				l.convDesc,
				l.bd_algo16,
				state.workspace,
				l.workspace_size,
				&beta,
				l.dsrcTensorDesc16,
				input16));    // state.delta);

			cuda_convert_f16_to_f32(input16, input16_size, state.delta);

			if (l.binary || l.xnor) swap_binary(&l);
			if (l.xnor) gradient_array_ongpu(original_input, l.batch*l.c*l.h*l.w, HARDTAN, state.delta);
		}
	}
	else {
		//#else    // CUDNN_HALF

		if(l.batch_normalize){
			backward_batchnorm_layer_gpu(l, state);
		}

		if (!state.net.adversarial && !l.train_only_bn) {

			float *old_input = state.input;

			// calculate conv weight updates
			// if used: beta=1 then loss decreases faster
			float one = 1.0f;
			CHECK_CUDNN(hipdnnConvolutionBackwardFilter(cudnn_handle(),
				&one,
				l.srcTensorDesc,
				state.input,
				l.ddstTensorDesc,
				l.delta_gpu,
				l.convDesc,
				l.bf_algo,
				state.workspace,
				l.workspace_size,
				&one,
				l.dweightDesc,
				l.weight_updates_gpu));

			state.input = old_input;
		}

		if (state.delta) {
			if (l.binary || l.xnor) swap_binary(&l);

			float *old_weights = l.weights_gpu;

			// http://docs.nvidia.com/deeplearning/sdk/cudnn-developer-guide/index.html#hipdnnConvolutionBackwardData
			// calculate delta for the next layer
			float one = 1.0f;
			CHECK_CUDNN(hipdnnConvolutionBackwardData(cudnn_handle(),
				&one,
				l.weightDesc,
				l.weights_gpu,
				l.ddstTensorDesc,
				l.delta_gpu,
				l.convDesc,
				l.bd_algo,
				state.workspace,
				l.workspace_size,
				&one,
				l.dsrcTensorDesc,
				state.delta));

			l.weights_gpu = old_weights;

			if (l.binary || l.xnor) swap_binary(&l);
			if (l.xnor) gradient_array_ongpu(original_input, l.batch*l.c*l.h*l.w, HARDTAN, state.delta);
		}
	}

//#endif    // CUDNN_HALF

#else    // CUDNN
	if (l.batch_normalize) {
		backward_batchnorm_layer_gpu(l, state);
	}

	int m = l.n / l.groups;
	int n = l.size*l.size*l.c / l.groups;
	int k = l.out_w*l.out_h;

	int i, j;
	for(i = 0; i < l.batch; ++i){
		for (j = 0; j < l.groups; ++j) {
			float * a = l.delta_gpu + (i*l.groups + j)*m*k;
			float * b = state.workspace;
			float * c = l.weight_updates_gpu + j*l.nweights / l.groups;

			float *im = state.input + (i*l.groups + j)*l.c / l.groups*l.h*l.w;

			if (!state.net.adversarial && !l.train_only_bn) {
				//im2col_ongpu(im, l.c / l.groups, l.h, l.w, l.size, l.stride, l.pad, state.workspace);
				im2col_gpu_ext(im,          // input
					l.c / l.groups,         // input channels
					l.h, l.w,               // input size (h, w)
					l.size, l.size,         // kernel size (h, w)
					l.pad * l.dilation, l.pad * l.dilation,   // padding (h, w)
					l.stride_y, l.stride_x,     // stride (h, w)
					l.dilation, l.dilation, // dilation (h, w)
					state.workspace);       // output
				//gemm_ongpu(0, 1, m, n, k, 1, a + i*m*k, k, b, k, 1, c, n);
				gemm_ongpu(0, 1, m, n, k, 1, a, k, b, k, 1, c, n);
			}

			if (state.delta) {
				if (l.binary || l.xnor) swap_binary(&l);
				float * a = l.weights_gpu + j*l.nweights / l.groups;
				float * b = l.delta_gpu + (i*l.groups + j)*m*k;
				float * c = state.workspace;

				//gemm_ongpu(1, 0, n, k, m, 1, a, n, b + i*k*m, k, 0, c, k);
				gemm_ongpu(1, 0, n, k, m, 1, a, n, b, k, 0, c, k);


				float *delta = state.delta + (i*l.groups + j)*l.c / l.groups*l.h*l.w;

				//col2im_ongpu(state.workspace, l.c / l.groups, l.h, l.w, l.size, l.stride, l.pad, delta);
				col2im_gpu_ext(
					state.workspace,        // input
					l.c / l.groups,         // input channels
					l.h, l.w,               // input size (h, w)
					l.size, l.size,         // kernel size (h, w)
					l.pad * l.dilation, l.pad * l.dilation,   // padding size (h, w)
					l.stride_y, l.stride_x,     // stride size (h, w)
					l.dilation, l.dilation, // dilation size (h, w)
					delta);                 // output (delta)

				if (l.binary || l.xnor) {
					swap_binary(&l);
				}
				if (l.xnor) gradient_array_ongpu(original_input + i*l.c*l.h*l.w, l.c*l.h*l.w, HARDTAN, state.delta + i*l.c*l.h*l.w);
			}
		}
	}
#endif
	if (state.net.try_fix_nan) {
		if (state.delta) {
			reset_nan_and_inf(state.delta, l.inputs * l.batch);
		}
		int size = l.nweights;
		reset_nan_and_inf(l.weight_updates_gpu, size);
		fix_nan_and_inf(l.weights_gpu, size);
	}


}

__global__ void calc_avg_activation_kernel(float *src, float *dst, int size, int channels, int batches)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int xy = i % size;
	int b = i / size;

	if (i < size*batches) {
		dst[i] = 0;
		for (int c = 0; c < channels; ++c) {
			dst[i] += src[xy + size*(c + channels*b)];
		}
		dst[i] = dst[i] / channels;
	}
}

void calc_avg_activation_gpu(float *src, float *dst, int size, int channels, int batches)
{
	TAT(TATPARMS);

	const int num_blocks = get_number_of_blocks(size*batches, BLOCK);

	calc_avg_activation_kernel <<<num_blocks, BLOCK, 0, get_cuda_stream() >>> (src, dst, size, channels, batches);
}


__global__ void assisted_activation_kernel(float alpha, float *output, float *gt_gpu, float *a_avg_gpu, int size, int channels, int batches)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int xy = i % size;
	int b = i / size;

	if (b < batches)
	{
		for (int c = 0; c < channels; ++c)
		{
			output[xy + size*(c + channels*b)] += alpha * gt_gpu[i] * a_avg_gpu[i];
		}
	}
}

void assisted_activation_gpu(float alpha, float *output, float *gt_gpu, float *a_avg_gpu, int size, int channels, int batches)
{
	TAT(TATPARMS);

	const int num_blocks = get_number_of_blocks(size*batches, BLOCK);

	assisted_activation_kernel <<<num_blocks, BLOCK, 0, get_cuda_stream() >>> (alpha, output, gt_gpu, a_avg_gpu, size, channels, batches);
}


__global__ void assisted_activation2_kernel(float alpha, float *output, float *gt_gpu, float *a_avg_gpu, int size, int channels, int batches)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int xy = i % size;
	int b = i / size;
	float beta = 1 - alpha;

	if (b < batches) {
		for (int c = 0; c < channels; ++c) {
			if(gt_gpu[i] == 0)
				output[xy + size*(c + channels*b)] *= beta;

		}
	}
}

void assisted_activation2_gpu(float alpha, float *output, float *gt_gpu, float *a_avg_gpu, int size, int channels, int batches)
{
	TAT(TATPARMS);

	const int num_blocks = get_number_of_blocks(size*batches, BLOCK);

	assisted_activation2_kernel <<<num_blocks, BLOCK, 0, get_cuda_stream() >>> (alpha, output, gt_gpu, a_avg_gpu, size, channels, batches);
}

void assisted_excitation_forward_gpu(Darknet::Layer & l, Darknet::NetworkState state)
{
	TAT(TATPARMS);

	const int iteration_num = get_current_iteration(state.net); //(*state.net.seen) / (state.net.batch*state.net.subdivisions);

	float alpha = (1 + cos(3.141592 * iteration_num / state.net.max_batches)) / 2;

	if (l.assisted_excitation == 1)
	{
		if (iteration_num > state.net.max_batches / 2) return;
	}
	else
	{
		if (iteration_num < state.net.burn_in) return;
		else
			if (iteration_num > l.assisted_excitation) return;
		else
			alpha = (1 + cos(3.141592 * iteration_num / (state.net.burn_in + l.assisted_excitation))) / 2; // from 1 to 0
	}

	float *a_avg = (float *)calloc(l.out_w * l.out_h * l.batch, sizeof(float));
	float *gt = (float *)calloc(l.out_w * l.out_h * l.batch, sizeof(float));

	int b;
	int w, h;

	l.max_boxes = state.net.num_boxes;
	l.truths = l.max_boxes*(4 + 1);

	int num_truth = l.batch*l.truths;
	float *truth_cpu = (float *)calloc(num_truth, sizeof(float));
	cuda_pull_array(state.truth, truth_cpu, num_truth);

	for (b = 0; b < l.batch; ++b)
	{
		// calculate G
		int t;
		for (t = 0; t < state.net.num_boxes; ++t) {
			Darknet::Box truth = float_to_box_stride(truth_cpu + t*(4 + 1) + b*l.truths, 1);
			if (!truth.x) break;  // continue;
			float beta = 0;
			//float beta = 1 - alpha; // from 0 to 1
			float dw = (1 - truth.w) * beta;
			float dh = (1 - truth.h) * beta;

			int left = floorf((truth.x - (dw + truth.w) / 2) * l.out_w);
			int right = ceilf((truth.x + (dw + truth.w) / 2) * l.out_w);
			int top = floorf((truth.y - (dh + truth.h) / 2) * l.out_h);
			int bottom = ceilf((truth.y + (dh + truth.h) / 2) * l.out_h);
			if (left < 0) left = 0;
			if (top < 0) top = 0;
			if (right > l.out_w) right = l.out_w;
			if (bottom > l.out_h) bottom = l.out_h;

			for (w = left; w <= right; w++) {
				for (h = top; h < bottom; h++) {
					gt[w + l.out_w * h + l.out_w*l.out_h*b] = 1;
				}
			}
		}
	}

	cuda_push_array(l.gt_gpu, gt, l.out_w * l.out_h * l.batch);

	// calc avg_output on GPU - for whole batch
	calc_avg_activation_gpu(l.output_gpu, l.a_avg_gpu, l.out_w * l.out_h, l.out_c, l.batch);

	// calc new output
	assisted_activation_gpu(alpha, l.output_gpu, l.gt_gpu, l.a_avg_gpu, l.out_w * l.out_h, l.out_c, l.batch);

	if (0)   // visualize ground truth
	{
		cuda_pull_array(l.output_gpu, l.output, l.outputs * l.batch);
		CHECK_CUDA(hipStreamSynchronize(get_cuda_stream()));

		for (b = 0; b < l.batch; ++b)
		{
			*cfg_and_state.output << "Assisted Excitation alpha = " << alpha << std::endl;
			Darknet::Image img = Darknet::float_to_image(l.out_w, l.out_h, 1, &gt[l.out_w*l.out_h*b]);
			char buff[100];
			sprintf(buff, "a_excitation_gt_%d", b);
			show_image_cv(img, buff);

			//image img2 = float_to_image(l.out_w, l.out_h, 1, &l.output[l.out_w*l.out_h*l.out_c*b]);
			Darknet::Image img2 = Darknet::float_to_image_scaled(l.out_w, l.out_h, 1, &l.output[l.out_w*l.out_h*l.out_c*b]);
			char buff2[100];
			sprintf(buff2, "a_excitation_output_%d", b);
			show_image_cv(img2, buff2);

			cv::waitKey(5);
		}
		cv::waitKey(0);
	}

	free(truth_cpu);
	free(gt);
	free(a_avg);
}

void pull_convolutional_layer(Darknet::Layer & l)
{
	TAT(TATPARMS);

	cuda_pull_array_async(l.weights_gpu, l.weights, l.nweights);
	cuda_pull_array_async(l.biases_gpu, l.biases, l.n);
	if (l.weight_updates_gpu) cuda_pull_array_async(l.weight_updates_gpu, l.weight_updates, l.nweights);
	if (l.bias_updates_gpu) cuda_pull_array_async(l.bias_updates_gpu, l.bias_updates, l.n);
	if (l.batch_normalize){
		cuda_pull_array_async(l.scales_gpu, l.scales, l.n);
		cuda_pull_array_async(l.rolling_mean_gpu, l.rolling_mean, l.n);
		cuda_pull_array_async(l.rolling_variance_gpu, l.rolling_variance, l.n);
	}
	if (l.adam){
		cuda_pull_array_async(l.m_gpu, l.m, l.nweights);
		cuda_pull_array_async(l.v_gpu, l.v, l.nweights);
	}
	CHECK_CUDA(hipPeekAtLastError());
	CHECK_CUDA(hipStreamSynchronize(get_cuda_stream()));
}

void push_convolutional_layer(Darknet::Layer & l)
{
	TAT(TATPARMS);

	cuda_push_array(l.weights_gpu, l.weights, l.nweights);
#ifdef CUDNN_HALF
	assert(l.nweights > 0);
	cuda_convert_f32_to_f16(l.weights_gpu, l.nweights, l.weights_gpu16);
#endif
	cuda_push_array(l.biases_gpu, l.biases, l.n);
	if (l.train) {
		cuda_push_array(l.weight_updates_gpu, l.weight_updates, l.nweights);
		cuda_push_array(l.bias_updates_gpu, l.bias_updates, l.n);
	}
	if (l.batch_normalize){
		cuda_push_array(l.scales_gpu, l.scales, l.n);
		cuda_push_array(l.rolling_mean_gpu, l.rolling_mean, l.n);
		cuda_push_array(l.rolling_variance_gpu, l.rolling_variance, l.n);
	}
	if (l.adam){
		cuda_push_array(l.m_gpu, l.m, l.nweights);
		cuda_push_array(l.v_gpu, l.v, l.nweights);
	}
	CHECK_CUDA(hipPeekAtLastError());
}

void update_convolutional_layer_gpu(Darknet::Layer & l, int batch, float learning_rate_init, float momentum, float decay, float loss_scale)
{
	TAT(TATPARMS);

	if (l.deform)
	{
		if (l.rotate) rotate_weights_gpu(l.weight_updates_gpu, l.weight_deform_gpu, l.nweights, l.n, l.size, 1);
		else if (l.sway) sway_and_flip_weights_gpu(l.weight_updates_gpu, l.weight_deform_gpu, l.nweights, l.n, l.size, l.angle, 1);
		else if (l.stretch) stretch_weights_gpu(l.weight_updates_gpu, l.weight_deform_gpu, l.nweights, l.n, l.size, 0, 1);
		else if (l.stretch_sway) stretch_sway_flip_weights_gpu(l.weight_updates_gpu, l.weight_deform_gpu, l.nweights, l.n, l.size, l.angle, 1);

		reduce_and_expand_array_gpu(l.weight_deform_gpu, l.weight_updates_gpu, l.nweights, 4);
	}

	// Loss scale for Mixed-Precision on Tensor-Cores
	float learning_rate = learning_rate_init*l.learning_rate_scale / loss_scale;

	reset_nan_and_inf(l.weight_updates_gpu, l.nweights);
	fix_nan_and_inf(l.weights_gpu, l.nweights);

	// Gradient Centralization
	if (l.grad_centr && l.batch_normalize)
	{
		gradient_centralization_gpu(l.size, l.size, l.c / l.groups, l.n, l.weight_updates_gpu);
	}

	if (l.adam)
	{
		adam_update_gpu(l.weights_gpu, l.weight_updates_gpu, l.m_gpu, l.v_gpu, l.B1, l.B2, l.eps, decay, learning_rate, l.nweights, batch, l.t);

		adam_update_gpu(l.biases_gpu, l.bias_updates_gpu, l.bias_m_gpu, l.bias_v_gpu, l.B1, l.B2, l.eps, decay, learning_rate, l.n, batch, l.t);
		if (l.scales_gpu)
		{
			adam_update_gpu(l.scales_gpu, l.scale_updates_gpu, l.scale_m_gpu, l.scale_v_gpu, l.B1, l.B2, l.eps, decay, learning_rate, l.n, batch, l.t);
		}
	}
	else
	{
		float *old_weight_updates_gpu = l.weight_updates_gpu;

		if (l.reverse)
		{
			float clip = 0.0;
			float divider = 1.0;
			float abs_add = 1.0;
			mult_inverse_array_gpu(l.weight_updates_gpu, l.output_gpu, l.inputs*l.batch, l.reverse, divider, clip, abs_add);
			l.weight_updates_gpu = l.output_gpu;
		}

		axpy_ongpu(l.nweights, -decay*batch*loss_scale, l.weights_gpu, 1, l.weight_updates_gpu, 1);
		axpy_ongpu(l.nweights, learning_rate / batch, l.weight_updates_gpu, 1, l.weights_gpu, 1);

		l.weight_updates_gpu = old_weight_updates_gpu;

		scal_ongpu(l.nweights, momentum, l.weight_updates_gpu, 1);

		axpy_ongpu(l.n, learning_rate / batch, l.bias_updates_gpu, 1, l.biases_gpu, 1);
		scal_ongpu(l.n, momentum, l.bias_updates_gpu, 1);

		if (l.scales_gpu) {
			axpy_ongpu(l.n, learning_rate / batch, l.scale_updates_gpu, 1, l.scales_gpu, 1);
			scal_ongpu(l.n, momentum, l.scale_updates_gpu, 1);
		}
	}

	if (l.deform)
	{
		expand_array_gpu(l.weights_gpu, l.weight_deform_gpu, l.nweights, 4);

		if (l.rotate) rotate_weights_gpu(l.weight_deform_gpu, l.weights_gpu, l.nweights, l.n, l.size, 0);
		else if (l.sway) sway_and_flip_weights_gpu(l.weight_deform_gpu, l.weights_gpu, l.nweights, l.n, l.size, l.angle, 0);
		else if (l.stretch) stretch_weights_gpu(l.weight_deform_gpu, l.weights_gpu, l.nweights, l.n, l.size, 0, 0);
		else if (l.stretch_sway) stretch_sway_flip_weights_gpu(l.weight_deform_gpu, l.weights_gpu, l.nweights, l.n, l.size, l.angle, 0);
	}

	if (l.clip)
	{
		constrain_ongpu(l.nweights, l.clip, l.weights_gpu, 1);
	}
}
