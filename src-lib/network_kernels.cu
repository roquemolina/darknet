#include "hip/hip_runtime.h"
#include "darknet_internal.hpp"


namespace
{
	static auto & cfg_and_state = Darknet::CfgAndState::get();
}


typedef struct time_benchmark_layers
{
	float time;
	int layer_id;
	Darknet::ELayerType layer_type;
} time_benchmark_layers;


int time_comparator(const void *pa, const void *pb)
{
	TAT(TATPARMS);

	time_benchmark_layers a = *(time_benchmark_layers *)pa;
	time_benchmark_layers b = *(time_benchmark_layers *)pb;
	float diff = a.time - b.time;
	if (diff < 0) return 1;
	else if (diff > 0) return -1;
	return 0;
}

void forward_network_gpu(Darknet::Network & net, Darknet::NetworkState state)
{
	TAT(TATPARMS);

	static time_benchmark_layers *avg_time_per_layer = NULL;
	static time_benchmark_layers *sorted_avg_time_per_layer = NULL;
	if (net.benchmark_layers)
	{
		if (!avg_time_per_layer)
		{
			avg_time_per_layer = (time_benchmark_layers *)calloc(net.n, sizeof(time_benchmark_layers));
			sorted_avg_time_per_layer = (time_benchmark_layers *)calloc(net.n, sizeof(time_benchmark_layers));
		}
		/// @todo in previous versions we did not CHECK_CUDA here -- was that intentional?
		CHECK_CUDA(hipDeviceSynchronize()); // was this removed in CUDA 11.6+?
	}

	state.workspace = net.workspace;
	for (int i = 0; i < net.n; ++i)
	{
		state.index = i;
		Darknet::Layer & l = net.layers[i];

		if (l.delta_gpu && state.train)
		{
			fill_ongpu(l.outputs * l.batch, 0, l.delta_gpu, 1);
		}

		l.forward_gpu(l, state);

		if(net.wait_stream)
		{
			CHECK_CUDA(hipStreamSynchronize(get_cuda_stream()));
		}
		state.input = l.output_gpu;
	}

	if (net.benchmark_layers)
	{
		*cfg_and_state.output << std::endl << std::endl << "Sorted by time (forward):" << std::endl;

		/// @todo replace qsort() low priority
		qsort(sorted_avg_time_per_layer, net.n, sizeof(time_benchmark_layers), time_comparator);

		for (int i = 0; i < net.n; ++i)
		{
			*cfg_and_state.output
				<< i
				<< " - fw-sort-layer " << sorted_avg_time_per_layer[i].layer_id
				<< " - type: " << static_cast<int>(sorted_avg_time_per_layer[i].layer_type)
				<< " - avg_time " << sorted_avg_time_per_layer[i].time << " ms"
				<< std::endl;
		}
	}
}

void backward_network_gpu(Darknet::Network & net, Darknet::NetworkState state)
{
	TAT(TATPARMS);

	static time_benchmark_layers *avg_time_per_layer = NULL;
	static time_benchmark_layers *sorted_avg_time_per_layer = NULL;
	if (net.benchmark_layers)
	{
		if (!avg_time_per_layer)
		{
			avg_time_per_layer = (time_benchmark_layers *)calloc(net.n, sizeof(time_benchmark_layers));
			sorted_avg_time_per_layer = (time_benchmark_layers *)calloc(net.n, sizeof(time_benchmark_layers));
		}
		CHECK_CUDA(hipDeviceSynchronize());
	}

	state.workspace = net.workspace;
	int i;
	float * original_input = state.input;
	float * original_delta = state.delta;
	for(i = net.n-1; i >= 0; --i)
	{
		state.index = i;
		Darknet::Layer & l = net.layers[i];
		if (l.stopbackward == 1)
		{
			break;
		}

		if (l.stopbackward > get_current_iteration(net))
		{
			break;
		}

		if (i == 0)
		{
			state.input = original_input;
			state.delta = original_delta;
		}
		else
		{
			const Darknet::Layer & prev = net.layers[i-1];
			state.input = prev.output_gpu;
			state.delta = prev.delta_gpu;
			if (net.optimized_memory && !prev.keep_delta_gpu)
			{
				state.delta = net.state_delta_gpu;
			}
		}

		if (l.onlyforward)
		{
			continue;
		}

		l.backward_gpu(l, state);

		if (i != 0)
		{
			Darknet::Layer & prev = net.layers[i - 1];
			if (net.optimized_memory && state.delta && !prev.keep_delta_gpu)
			{
				if (prev.delta_gpu != state.delta)
				{
					simple_copy_ongpu(prev.outputs*prev.batch, state.delta, prev.delta_gpu);
				}
				fill_ongpu(prev.outputs*prev.batch, 0, net.state_delta_gpu, 1);
			}
		}
	}

	if (net.adversarial && net.attention)
	{
		int img_size = net.w * net.h * net.c;
		float *original_input_cpu = (float *)xcalloc(img_size, sizeof(float));
		float *original_delta_cpu = (float *)xcalloc(img_size, sizeof(float));
		cuda_pull_array(original_input, original_input_cpu, img_size);
		cuda_pull_array(original_delta, original_delta_cpu, img_size);

		Darknet::Image attention_img = Darknet::make_attention_image(img_size, original_delta_cpu, original_input_cpu, net.w, net.h, net.c, 0.7);
		Darknet::show_image(attention_img, "attention_img");
		cv::resizeWindow("attention_img", 500, 500);

		Darknet::free_image(attention_img);

		Darknet::Image attention_mask_img = Darknet::make_attention_image(img_size, original_delta_cpu, original_delta_cpu, net.w, net.h, net.c, 1.0);
		Darknet::show_image(attention_mask_img, "attention_mask_img");
		cv::resizeWindow("attention_mask_img", 500, 500);

		Darknet::free_image(attention_mask_img);

		free(original_input_cpu);
		free(original_delta_cpu);
	}

	if (net.adversarial)
	{
		int x_size = get_network_input_size(net) * net.batch;
		*cfg_and_state.output
			<< "x_size=" << x_size
			<< ", original_delta=" << original_delta
			<< ", original_input=" << original_input
			<< ", net.learning_rate=" << net.learning_rate
			<< std::endl;
		axpy_ongpu(x_size, net.learning_rate, original_delta, 1, original_input, 1);
		constrain_min_max_ongpu(x_size, 0, 1, original_input, 1);
	}

	if (net.benchmark_layers)
	{
		*cfg_and_state.output << std::endl << std::endl << "Sorted by time (backward):" << std::endl;

		/// @todo replace qsort() unknown priority
		qsort(sorted_avg_time_per_layer, net.n, sizeof(time_benchmark_layers), time_comparator);

		for (i = 0; i < net.n; ++i)
		{
			*cfg_and_state.output
				<< i
				<< " - bw-sort-layer " << sorted_avg_time_per_layer[i].layer_id
				<< " - type: " << static_cast<int>(sorted_avg_time_per_layer[i].layer_type)
				<< " - avg_time " << sorted_avg_time_per_layer[i].time << " ms"
				<< std::endl;
		}
	}
}

void update_network_gpu(Darknet::Network & net)
{
	TAT(TATPARMS);

	cuda_set_device(net.gpu_index);
	const int iteration_num = (*net.seen) / (net.batch * net.subdivisions);

	int update_batch = net.batch*net.subdivisions * get_sequence_value(net);

	float rate = get_current_rate(net);
	for (int i = 0; i < net.n; ++i)
	{
		Darknet::Layer & l = net.layers[i];
		if (l.train == 0)
		{
			continue;
		}
		l.t = get_current_batch(net);
		if (iteration_num > (net.max_batches * 1 / 2))
		{
			l.deform = 0;
		}
		if (l.burnin_update && (l.burnin_update*net.burn_in > iteration_num))
		{
			continue;
		}
		if (l.train_only_bn)
		{
			continue;
		}

		if (l.update_gpu && l.dont_update < iteration_num)
		{
			l.update_gpu(l, update_batch, rate, net.momentum, net.decay, net.loss_scale);
		}
	}
}

void forward_backward_network_gpu(Darknet::Network & net, float *x, float *y)
{
	TAT(TATPARMS);

	Darknet::NetworkState state;
	state.index = 0;
	state.net = net;
	int x_size = get_network_input_size(net)*net.batch;
	int y_size = get_network_output_size(net)*net.batch;
	if (net.layers[net.n-1].truths)
	{
		y_size = net.layers[net.n-1].truths*net.batch;
	}
	if (!*net.input_gpu)
	{
		*net.input_gpu = cuda_make_array(x, x_size);
		*net.truth_gpu = cuda_make_array(y, y_size);
	}
	else
	{
		cuda_push_array(*net.input_gpu, x, x_size);
		cuda_push_array(*net.truth_gpu, y, y_size);
	}
	state.input = *net.input_gpu;
	state.delta = 0;
	if (net.adversarial)
	{
		state.delta = cuda_make_array(NULL, x_size);
	}
	state.truth = *net.truth_gpu;
	state.train = 1;
#if defined(CUDNN_HALF) && defined(CUDNN)
	int i;
	for (i = 0; i < net.n; ++i)
	{
		Darknet::Layer & l = net.layers[i];
		if (net.cudnn_half)
		{
			if (l.type == Darknet::ELayerType::CONVOLUTIONAL && l.weights_gpu && l.weights_gpu16)
			{
				assert((l.nweights) > 0);
				cuda_convert_f32_to_f16(l.weights_gpu, l.nweights, l.weights_gpu16);
			}
			else if (l.type == Darknet::ELayerType::CRNN && l.input_layer->weights_gpu && l.input_layer->weights_gpu16)
			{
				assert((l.input_layer->c*l.input_layer->n*l.input_layer->size*l.input_layer->size) > 0);
				cuda_convert_f32_to_f16(l.input_layer->weights_gpu, l.input_layer->nweights, l.input_layer->weights_gpu16);
				cuda_convert_f32_to_f16(l.self_layer->weights_gpu, l.self_layer->nweights, l.self_layer->weights_gpu16);
				cuda_convert_f32_to_f16(l.output_layer->weights_gpu, l.output_layer->nweights, l.output_layer->weights_gpu16);
			}
		}
	}
#endif
	forward_network_gpu(net, state);
	//hipStreamSynchronize(get_cuda_stream());
	backward_network_gpu(net, state);

	if (net.adversarial)
	{
		cuda_free(state.delta);
		cuda_pull_array(*net.input_gpu, x, x_size);
	}
}

float train_network_datum_gpu(Darknet::Network & net, float *x, float *y)
{
	TAT(TATPARMS);

	*net.seen += net.batch;
	if (net.adversarial_lr && rand_int(0, 1) == 1 && get_current_iteration(net) > net.burn_in)
	{
		net.adversarial = 1;
		float lr_old = net.learning_rate;
		float scale = (get_current_iteration(net) / ((float)net.max_batches));
		//scale = sin(scale * M_PI);
		net.learning_rate = net.adversarial_lr * scale;
		int y_size = get_network_output_size(net)*net.batch;
		if (net.layers[net.n - 1].truths)
		{
			y_size = net.layers[net.n - 1].truths*net.batch;
		}
		float *truth_cpu = (float *)xcalloc(y_size, sizeof(float));

		const int img_size = net.w*net.h*net.c;
		float *old_input = (float *)xcalloc(img_size*net.batch, sizeof(float));
		memcpy(old_input, x, img_size*net.batch * sizeof(float));

		*cfg_and_state.output << std::endl << "adversarial training, adversarial_lr=" << net.adversarial_lr * scale << std::endl;

		forward_backward_network_gpu(net, x, truth_cpu);

		int b;
		for (b = 0; b < net.batch; ++b)
		{
			if (b % 2 == 1 && net.contrastive)
			{
				memcpy(x + img_size*b, old_input + img_size*b, img_size * sizeof(float));
			}
		}

		Darknet::Image im;
		im.w = net.w;
		im.h = net.h;
		im.c = net.c;
		im.data = x;
		Darknet::show_image(im, "adversarial data augmentation");
		cv::resizeWindow("adversarial data augmentation", 500, 500);
		cv::waitKey(1);

		free(old_input);
		free(truth_cpu);
		net.learning_rate = lr_old;
		net.adversarial = 0;
	}
	forward_backward_network_gpu(net, x, y);
	float error = get_network_cost(net);

	return error;
}


void pull_updates(Darknet::Layer & l)
{
	TAT(TATPARMS);

	if (l.type == Darknet::ELayerType::CONVOLUTIONAL)
	{
		cuda_pull_array(l.bias_updates_gpu, l.bias_updates, l.n);
		cuda_pull_array(l.weight_updates_gpu, l.weight_updates, l.nweights);
		if(l.scale_updates)
		{
			cuda_pull_array(l.scale_updates_gpu, l.scale_updates, l.n);
		}
	}
	else if (l.type == Darknet::ELayerType::CONNECTED)
	{
		cuda_pull_array(l.bias_updates_gpu, l.bias_updates, l.outputs);
		cuda_pull_array(l.weight_updates_gpu, l.weight_updates, l.outputs*l.inputs);
	}
}

void push_updates(Darknet::Layer & l)
{
	TAT(TATPARMS);

	if (l.type == Darknet::ELayerType::CONVOLUTIONAL)
	{
		cuda_push_array(l.bias_updates_gpu, l.bias_updates, l.n);
		cuda_push_array(l.weight_updates_gpu, l.weight_updates, l.nweights);
		if(l.scale_updates) cuda_push_array(l.scale_updates_gpu, l.scale_updates, l.n);
	}
	else if (l.type == Darknet::ELayerType::CONNECTED)
	{
		cuda_push_array(l.bias_updates_gpu, l.bias_updates, l.outputs);
		cuda_push_array(l.weight_updates_gpu, l.weight_updates, l.outputs*l.inputs);
	}
}

void update_layer(Darknet::Layer & l, Darknet::Network net)
{
	TAT(TATPARMS);

	int update_batch = net.batch*net.subdivisions;
	float rate = get_current_rate(net);
	l.t = get_current_batch(net);
	if(l.update_gpu)
	{
		l.update_gpu(l, update_batch, rate, net.momentum, net.decay, net.loss_scale);
	}
}

void merge_weights(Darknet::Layer & l, Darknet::Layer & base)
{
	TAT(TATPARMS);

	if (l.type == Darknet::ELayerType::CONVOLUTIONAL)
	{
		axpy_cpu(l.n, 1, l.biases, 1, base.biases, 1);
		axpy_cpu(l.nweights, 1, l.weights, 1, base.weights, 1);
		if (l.scales)
		{
			axpy_cpu(l.n, 1, l.scales, 1, base.scales, 1);
		}
	}
	else if (l.type == Darknet::ELayerType::CONNECTED)
	{
		axpy_cpu(l.outputs, 1, l.biases, 1, base.biases, 1);
		axpy_cpu(l.outputs*l.inputs, 1, l.weights, 1, base.weights, 1);
	}
}

void scale_weights(Darknet::Layer & l, float s)
{
	TAT(TATPARMS);

	if (l.type == Darknet::ELayerType::CONVOLUTIONAL)
	{
		scal_cpu(l.n, s, l.biases, 1);
		scal_cpu(l.nweights, s, l.weights, 1);
		if (l.scales)
		{
			scal_cpu(l.n, s, l.scales, 1);
		}
	}
	else if (l.type == Darknet::ELayerType::CONNECTED)
	{
		scal_cpu(l.outputs, s, l.biases, 1);
		scal_cpu(l.outputs*l.inputs, s, l.weights, 1);
	}
}


void pull_weights(Darknet::Layer & l)
{
	TAT(TATPARMS);

	if (l.type == Darknet::ELayerType::CONVOLUTIONAL)
	{
		cuda_pull_array(l.biases_gpu, l.biases, l.n);
		cuda_pull_array(l.weights_gpu, l.weights, l.nweights);
		if (l.scales)
		{
			cuda_pull_array(l.scales_gpu, l.scales, l.n);
		}
	}
	else if (l.type == Darknet::ELayerType::CONNECTED)
	{
		cuda_pull_array(l.biases_gpu, l.biases, l.outputs);
		cuda_pull_array(l.weights_gpu, l.weights, l.outputs*l.inputs);
	}
}

void push_weights(Darknet::Layer & l)
{
	TAT(TATPARMS);

	if(l.type == Darknet::ELayerType::CONVOLUTIONAL)
	{
		cuda_push_array(l.biases_gpu, l.biases, l.n);
		cuda_push_array(l.weights_gpu, l.weights, l.nweights);
		if(l.scales)
		{
			cuda_push_array(l.scales_gpu, l.scales, l.n);
		}
	}
	else if(l.type == Darknet::ELayerType::CONNECTED)
	{
		cuda_push_array(l.biases_gpu, l.biases, l.outputs);
		cuda_push_array(l.weights_gpu, l.weights, l.outputs*l.inputs);
	}
}

void distribute_weights(Darknet::Layer & l, Darknet::Layer & base)
{
	TAT(TATPARMS);

	if(l.type == Darknet::ELayerType::CONVOLUTIONAL)
	{
		cuda_push_array(l.biases_gpu, base.biases, l.n);
		cuda_push_array(l.weights_gpu, base.weights, l.nweights);
		if(base.scales) cuda_push_array(l.scales_gpu, base.scales, l.n);
	}
	else if(l.type == Darknet::ELayerType::CONNECTED)
	{
		cuda_push_array(l.biases_gpu, base.biases, l.outputs);
		cuda_push_array(l.weights_gpu, base.weights, l.outputs*l.inputs);
	}
}


void merge_updates(Darknet::Layer & l, Darknet::Layer & base)
{
	TAT(TATPARMS);

	if (l.type == Darknet::ELayerType::CONVOLUTIONAL) {
		axpy_cpu(l.n, 1, l.bias_updates, 1, base.bias_updates, 1);
		axpy_cpu(l.nweights, 1, l.weight_updates, 1, base.weight_updates, 1);
		if (l.scale_updates) {
			axpy_cpu(l.n, 1, l.scale_updates, 1, base.scale_updates, 1);
		}
	} else if(l.type == Darknet::ELayerType::CONNECTED) {
		axpy_cpu(l.outputs, 1, l.bias_updates, 1, base.bias_updates, 1);
		axpy_cpu(l.outputs*l.inputs, 1, l.weight_updates, 1, base.weight_updates, 1);
	}
}

void distribute_updates(Darknet::Layer & l, Darknet::Layer & base)
{
	TAT(TATPARMS);

	if(l.type == Darknet::ELayerType::CONVOLUTIONAL)
	{
		cuda_push_array(l.bias_updates_gpu, base.bias_updates, l.n);
		cuda_push_array(l.weight_updates_gpu, base.weight_updates, l.nweights);
		if(base.scale_updates)
		{
			cuda_push_array(l.scale_updates_gpu, base.scale_updates, l.n);
		}
	}
	else if (l.type == Darknet::ELayerType::CONNECTED)
	{
		cuda_push_array(l.bias_updates_gpu, base.bias_updates, l.outputs);
		cuda_push_array(l.weight_updates_gpu, base.weight_updates, l.outputs*l.inputs);
	}
}

void sync_layer(Darknet::Network * nets, int n, int j)
{
	TAT(TATPARMS);

	Darknet::Network net = nets[0];
	Darknet::Layer & base = net.layers[j];
	cuda_set_device(net.gpu_index);
	pull_weights(base);

	for (int i = 1; i < n; ++i)
	{
		cuda_set_device(nets[i].gpu_index);
		Darknet::Layer & l = nets[i].layers[j];
		pull_weights(l);
		merge_weights(l, base);
	}

	scale_weights(base, 1./n);

	for (int i = 0; i < n; ++i)
	{
		cuda_set_device(nets[i].gpu_index);
		Darknet::Layer & l = nets[i].layers[j];
		distribute_weights(l, base);
	}
}


void sync_nets(Darknet::Network * nets, int n, int interval)
{
	TAT(TATPARMS);

	int layers = nets[0].n;

	std::vector<std::thread> threads;
	threads.reserve(layers);

	*nets[0].seen += interval * (n-1) * nets[0].batch * nets[0].subdivisions;
	for (int j = 0; j < n; ++j)
	{
		*nets[j].seen = *nets[0].seen;
	}

	for (int j = 0; j < layers; ++j)
	{
		threads.emplace_back(
				[nets,n,j]()
				{
					sync_layer(nets, n, j);
				});
	}

	for (auto & t : threads)
	{
		t.join();
	}

	return;
}

float train_networks(Darknet::Network * nets, int n, data d, int interval)
{
	TAT(TATPARMS);

	// IMPORTANT:  If we get here, we already know that n > 1!  This is only called when we have multiple GPUs.
	// There is another similar function called train_network() for single GPU (note singular name!)

#ifdef _DEBUG
	int batch = nets[0].batch;
	int subdivisions = nets[0].subdivisions;
	assert(batch * subdivisions * n == d.X.rows);
#endif

	// "errors"?  This is "loss", right?  We're adding up the loss from training a batch on each GPU?
	float * errors = (float*) calloc(n, sizeof(float));

	std::vector<std::thread> threads;
	threads.reserve(n);
	std::vector<data> p(n);

	for (int i = 0; i < n; ++i)
	{
		 p[i] = get_data_part(d, i, n);

		threads.emplace_back(
			[](Darknet::Network & net, data &d2, float * err)
			{
				TAT(TATPARMS);

				cuda_set_device(net.gpu_index);
				*err = train_network(net, d2); // note this is the "singular" train function (e.g., for a single GPU)
			},
			std::ref(nets[i]), std::ref(p[i]), errors + i);
	}

	float sum = 0.0f;
	for (int i = 0; i < n; ++i)
	{
		threads[i].join();
		sum += errors[i];
	}
	free(errors);

	//hipDeviceSynchronize();
	*nets[0].cur_iteration += (n - 1);
	*nets[0].seen = nets[0].batch * nets[0].subdivisions * get_current_iteration(nets[0]); // remove this line, when you will save to weights-file both: seen & cur_iteration
	if (get_current_iteration(nets[0]) % interval == 0)
	{
		if (cfg_and_state.is_verbose)
		{
			*cfg_and_state.output << "Syncing..." << std::flush;
		}
		sync_nets(nets, n, interval);
		if (cfg_and_state.is_verbose)
		{
			*cfg_and_state.output << "done!" << std::endl;
		}
	}

	//hipDeviceSynchronize();
	return sum / n;
}

float *get_network_output_layer_gpu(Darknet::Network & net, int i)
{
	TAT(TATPARMS);

	Darknet::Layer & l = net.layers[i];
	if (l.type != Darknet::ELayerType::REGION && l.type != Darknet::ELayerType::YOLO && (*net.cuda_graph_ready) == 0)
	{
		cuda_pull_array(l.output_gpu, l.output, l.outputs*l.batch);
	}

	return l.output;
}

float *get_network_output_gpu(Darknet::Network & net)
{
	TAT(TATPARMS);

	int i;
	for (i = net.n - 1; i > 0; --i)
	{
		if (net.layers[i].type != Darknet::ELayerType::COST)
		{
			break;
		}
	}

	return get_network_output_layer_gpu(net, i);
}

float *network_predict_gpu(Darknet::Network & net, float *input)
{
	TAT(TATPARMS);

	if (net.gpu_index != cuda_get_device())
	{
		cuda_set_device(net.gpu_index);
	}
	int size = get_network_input_size(net) * net.batch;
	Darknet::NetworkState state;
	state.index = 0;
	state.net = net;
	//state.input = cuda_make_array(input, size);   // memory will be allocated in the parse_network_cfg_custom()
	state.input = net.input_state_gpu;
	memcpy(net.input_pinned_cpu, input, size * sizeof(float));
	state.truth = 0;
	state.train = 0;
	state.delta = 0;

	//hipGraphExec_t instance = (hipGraphExec_t)net.cuda_graph_exec;
	static hipGraphExec_t instance;

	if ((*net.cuda_graph_ready) == 0)
	{
		static hipGraph_t graph;
		if (net.use_cuda_graph == 1)
		{
			for (int i = 0; i < 16; ++i)
			{
				switch_stream(i);
			}

			hipStream_t stream0 = switch_stream(0);
			CHECK_CUDA(hipDeviceSynchronize());
			*cfg_and_state.output << "Try to capture graph..." << std::endl;
			//hipGraph_t graph = (hipGraph_t)net.cuda_graph;
			CHECK_CUDA(hipStreamBeginCapture(stream0, hipStreamCaptureModeGlobal));
		}

		cuda_push_array(state.input, net.input_pinned_cpu, size);
		forward_network_gpu(net, state);

		if (net.use_cuda_graph == 1)
		{
			hipStream_t stream0 = switch_stream(0);
			CHECK_CUDA(hipStreamEndCapture(stream0, &graph));
			CHECK_CUDA(hipGraphInstantiate(&instance, graph, NULL, NULL, 0));
			(*net.cuda_graph_ready) = 1;
			*cfg_and_state.output << "Graph is captured..." << std::endl;
			CHECK_CUDA(hipDeviceSynchronize());
		}

		CHECK_CUDA(hipStreamSynchronize(get_cuda_stream()));
	}
	else
	{
		hipStream_t stream0 = switch_stream(0);
		CHECK_CUDA( hipGraphLaunch(instance, stream0) );
		CHECK_CUDA( hipStreamSynchronize(stream0) );
	}

	float *out = get_network_output_gpu(net);
	reset_wait_stream_events();
	//cuda_free(state.input);   // will be freed in the free_network()
	return out;
}
