#include "hip/hip_runtime.h"
#include "darknet_internal.hpp"


__global__ void forward_avgpool_layer_kernel(int n, int w, int h, int c, float *input, float *output)
{
	int id = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
	if(id >= n) return;

	int k = id % c;
	id /= c;
	int b = id;

	int i;
	int out_index = (k + c*b);
	output[out_index] = 0;
	for(i = 0; i < w*h; ++i){
		int in_index = i + h*w*(k + b*c);
		output[out_index] += input[in_index];
	}
	output[out_index] /= w*h;
}

__global__ void backward_avgpool_layer_kernel(int n, int w, int h, int c, float *in_delta, float *out_delta)
{
	int id = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
	if(id >= n) return;

	int k = id % c;
	id /= c;
	int b = id;

	int i;
	int out_index = (k + c*b);
	for(i = 0; i < w*h; ++i){
		int in_index = i + h*w*(k + b*c);
		in_delta[in_index] += out_delta[out_index] / (w*h);
	}
}

void forward_avgpool_layer_gpu(Darknet::Layer & l, Darknet::NetworkState state)
{
	TAT(TATPARMS);

	size_t n = l.c * l.batch;

	forward_avgpool_layer_kernel<<<cuda_gridsize(n), BLOCK, 0, get_cuda_stream() >>>(n, l.w, l.h, l.c, state.input, l.output_gpu);
	CHECK_CUDA(hipPeekAtLastError());
}

void backward_avgpool_layer_gpu(Darknet::Layer & l, Darknet::NetworkState state)
{
	TAT(TATPARMS);

	size_t n = l.c * l.batch;

	backward_avgpool_layer_kernel<<<cuda_gridsize(n), BLOCK, 0, get_cuda_stream() >>>(n, l.w, l.h, l.c, state.delta, l.delta_gpu);
	CHECK_CUDA(hipPeekAtLastError());
}
