#include "hip/hip_runtime.h"
#include "darknet_internal.hpp"


__global__ void forward_maxpool_depth_layer_kernel(int n, int w, int h, int c, int out_c, int batch, float *input, float *output, int *indexes)
{
	int id = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
	if (id >= n) return;

	int j = id % w;
	id = id / w;
	int i = id % h;
	id = id / h;
	//int g = id % out_c;
	//id = id / out_c;
	int b = id % batch;

	int k;
	for (int g = 0; g < out_c; ++g)
	{
		int out_index = j + w*(i + h*(g + out_c*b));
		float max = -FLT_MAX;
		int max_i = -1;

		for (k = g; k < c; k += out_c)
		{
			int in_index = j + w*(i + h*(k + c*b));
			float val = input[in_index];

			max_i = (val > max) ? in_index : max_i;
			max = (val > max) ? val : max;
		}
		output[out_index] = max;
		if (indexes) indexes[out_index] = max_i;
	}
}


__global__ void backward_maxpool_depth_layer_kernel(int n, int w, int h, int c, int batch, float *delta, float *prev_delta, int *indexes)
{
	int id = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
	if (id >= n) return;

	int index = indexes[id];
	prev_delta[index] += delta[id];
}


__global__ void forward_maxpool_layer_kernel(int n, int in_h, int in_w, int in_c, int stride_x, int stride_y, int size, int pad, float *input, float *output, int *indexes)
{
	int h = (in_h + pad - size) / stride_y + 1;
	int w = (in_w + pad - size) / stride_x + 1;
	int c = in_c;

	int id = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
	if(id >= n) return;

	int j = id % w;
	id /= w;
	int i = id % h;
	id /= h;
	int k = id % c;
	id /= c;
	int b = id;

	int w_offset = -pad / 2;
	int h_offset = -pad / 2;

	int out_index = j + w*(i + h*(k + c*b));
	float max = -INFINITY;
	int max_i = -1;
	int l, m;
	for(l = 0; l < size; ++l){
		for(m = 0; m < size; ++m){
			int cur_h = h_offset + i*stride_y + l;
			int cur_w = w_offset + j*stride_x + m;
			int index = cur_w + in_w*(cur_h + in_h*(k + b*in_c));
			int valid = (cur_h >= 0 && cur_h < in_h &&
					cur_w >= 0 && cur_w < in_w);
			float val = (valid != 0) ? input[index] : -INFINITY;
			max_i = (val > max) ? index : max_i;
			max   = (val > max) ? val   : max;
		}
	}
	output[out_index] = max;
	if (indexes) indexes[out_index] = max_i;
}

__global__ void forward_zero_nonmax_kernel(int n, float *input, float *output)
{

	int id = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
	if (id >= n) return;

	if (input[id] != output[id]) output[id] = 0;
}

__global__ void backward_maxpool_layer_kernel(int n, int in_h, int in_w, int in_c, int stride_x, int stride_y, int size, int pad, float *delta, float *prev_delta, int *indexes)
{
	int h = (in_h + pad - size) / stride_y + 1;
	int w = (in_w + pad - size) / stride_x + 1;
	int c = in_c;
	int area_x = (size - 1) / stride_x;
	int area_y = (size - 1) / stride_y;

	int id = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
	if(id >= n) return;

	int index = id;
	int j = id % in_w;
	id /= in_w;
	int i = id % in_h;
	id /= in_h;
	int k = id % in_c;
	id /= in_c;
	int b = id;

	int w_offset = -pad / 2;
	int h_offset = -pad / 2;

	float d = 0;
	int l, m;
	for(l = -area_y; l < area_y+1; ++l){
		for(m = -area_x; m < area_x+1; ++m){
			int out_w = (j-w_offset)/stride_x + m;
			int out_h = (i-h_offset)/stride_y + l;
			int out_index = out_w + w*(out_h + h*(k + c*b));
			int valid = (out_w >= 0 && out_w < w &&
					out_h >= 0 && out_h < h);
			d += (valid && indexes[out_index] == index) ? delta[out_index] : 0;
		}
	}
	prev_delta[index] += d;
}

__global__ void backward_zero_nonmax_kernel(int n, int *indexes, float *prev_delta)
{

	int id = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
	if (id >= n) return;

	if (indexes[id] != id) prev_delta[id] = 0;
}

void forward_maxpool_layer_gpu(Darknet::Layer & l, Darknet::NetworkState state)
{
	TAT(TATPARMS);

	if (l.maxpool_depth)
	{
		int h = l.out_h;
		int w = l.out_w;
		int c = 1;// layer.out_c;

		size_t n = h*w*c*l.batch;

		forward_maxpool_depth_layer_kernel <<<cuda_gridsize(n), BLOCK, 0, get_cuda_stream() >>>(n, l.w, l.h, l.c, l.out_c, l.batch, state.input, l.output_gpu, l.indexes_gpu);
		CHECK_CUDA(hipPeekAtLastError());

		return;
	}

#ifdef CUDNN_DISABLED
	if (!state.train && l.stride == l.size)
	{
		// hipdnnPoolingBackward
		hipdnnStatus_t maxpool_status;

		float alpha = 1, beta = 0;
		maxpool_status = hipdnnPoolingForward(
			cudnn_handle(),
			l.poolingDesc,
			&alpha,
			l.srcTensorDesc,
			state.input,
			&beta,
			l.dstTensorDesc,
			l.output_gpu);

		//maxpool_status = hipdnnDestroyPoolingDescriptor(poolingDesc);
		//hipdnnDestroyTensorDescriptor(l.srcTensorDesc);
		//hipdnnDestroyTensorDescriptor(l.dstTensorDesc);
	}
	else
#endif
	{
		int h = l.out_h;
		int w = l.out_w;
		int c = l.out_c;

		size_t n = h * w * c * l.batch;

		forward_maxpool_layer_kernel <<<cuda_gridsize(n), BLOCK, 0, get_cuda_stream() >>> (n, l.h, l.w, l.c, l.stride_x, l.stride_y, l.size, l.pad, state.input, l.output_gpu, l.indexes_gpu);
		CHECK_CUDA(hipPeekAtLastError());

		if (l.maxpool_zero_nonmax)
		{
			forward_zero_nonmax_kernel <<<cuda_gridsize(n), BLOCK, 0, get_cuda_stream() >>> (n, state.input, l.output_gpu);
			CHECK_CUDA(hipPeekAtLastError());
		}
	}

	if (l.antialiasing)
	{
		Darknet::NetworkState s = { 0 };
		s.train = state.train;
		s.workspace = state.workspace;
		s.net = state.net;
		if (!state.train) s.index = state.index;  // don't use TC for training (especially without cuda_convert_f32_to_f16() )
		s.input = l.output_gpu;
		forward_convolutional_layer_gpu(*(l.input_layer), s);
		simple_copy_ongpu(l.outputs*l.batch, l.output_gpu, l.input_antialiasing_gpu);
		simple_copy_ongpu(l.input_layer->outputs*l.input_layer->batch, l.input_layer->output_gpu, l.output_gpu);
	}
}

void backward_maxpool_layer_gpu(Darknet::Layer & l, Darknet::NetworkState state)
{
	TAT(TATPARMS);

	if (l.antialiasing)
	{
		Darknet::NetworkState s = { 0 };
		s.train = state.train;
		s.workspace = state.workspace;
		s.net = state.net;
		s.delta = l.delta_gpu;  // s.delta will be returned to l.delta_gpu
		s.input = l.input_antialiasing_gpu;
		//if (!state.train) s.index = state.index;  // don't use TC for training (especially without cuda_convert_f32_to_f16() )
		simple_copy_ongpu(l.input_layer->outputs*l.input_layer->batch, l.delta_gpu, l.input_layer->delta_gpu);
		backward_convolutional_layer_gpu(*(l.input_layer), s);

		//simple_copy_ongpu(l.outputs*l.batch, l.input_antialiasing_gpu, l.output_gpu);
	}

	if (l.maxpool_depth)
	{
		int h = l.out_h;
		int w = l.out_w;
		int c = l.out_c;

		size_t n = h * w * c * l.batch;

		backward_maxpool_depth_layer_kernel <<<cuda_gridsize(n), BLOCK, 0, get_cuda_stream() >>>(n, l.w, l.h, l.c, l.batch, l.delta_gpu, state.delta, l.indexes_gpu);
		CHECK_CUDA(hipPeekAtLastError());
		return;
	}

	size_t n = l.h*l.w*l.c*l.batch;

	backward_maxpool_layer_kernel<<<cuda_gridsize(n), BLOCK, 0, get_cuda_stream() >>>(n, l.h, l.w, l.c, l.stride_x, l.stride_y, l.size, l.pad, l.delta_gpu, state.delta, l.indexes_gpu);
	CHECK_CUDA(hipPeekAtLastError());

	if (l.maxpool_zero_nonmax)
	{
		backward_zero_nonmax_kernel <<<cuda_gridsize(n), BLOCK, 0, get_cuda_stream() >>> (n, l.indexes_gpu, state.delta);
		CHECK_CUDA(hipPeekAtLastError());
	}
}




__global__ void forward_local_avgpool_layer_kernel(int n, int in_h, int in_w, int in_c, int stride_x, int stride_y, int size, int pad, float *input, float *output)
{
	int h = (in_h + pad - size) / stride_y + 1;
	int w = (in_w + pad - size) / stride_x + 1;
	int c = in_c;

	int id = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
	if (id >= n) return;

	int j = id % w;
	id /= w;
	int i = id % h;
	id /= h;
	int k = id % c;
	id /= c;
	int b = id;

	int w_offset = -pad / 2;
	int h_offset = -pad / 2;

	int out_index = j + w*(i + h*(k + c*b));
	float avg = 0;
	int counter = 0;
	int l, m;
	for (l = 0; l < size; ++l) {
		for (m = 0; m < size; ++m) {
			int cur_h = h_offset + i*stride_y + l;
			int cur_w = w_offset + j*stride_x + m;
			int index = cur_w + in_w*(cur_h + in_h*(k + b*in_c));
			int valid = (cur_h >= 0 && cur_h < in_h &&
				cur_w >= 0 && cur_w < in_w);
			if (valid) {
				counter++;
				avg += input[index];
			}
		}
	}
	output[out_index] = avg / counter;  // as HIPDNN_POOLING_AVERAGE_COUNT_EXCLUDE_PADDING
}


__global__ void backward_local_avgpool_layer_kernel(int n, int in_h, int in_w, int in_c, int stride_x, int stride_y, int size, int pad, float *delta, float *prev_delta)
{
	int h = (in_h + pad - size) / stride_y + 1;
	int w = (in_w + pad - size) / stride_x + 1;
	int c = in_c;
	int area_x = (size - 1) / stride_x;
	int area_y = (size - 1) / stride_y;

	int id = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
	if (id >= n) return;

	int index = id;
	int j = id % in_w;
	id /= in_w;
	int i = id % in_h;
	id /= in_h;
	int k = id % in_c;
	id /= in_c;
	int b = id;

	int w_offset = -pad / 2;
	int h_offset = -pad / 2;

	int counter = 0;
	float d = 0;
	int l, m;
	for (l = -area_y; l < area_y + 1; ++l) {
		for (m = -area_x; m < area_x + 1; ++m) {
			int out_w = (j - w_offset) / stride_x + m;
			int out_h = (i - h_offset) / stride_y + l;
			int out_index = out_w + w*(out_h + h*(k + c*b));
			int valid = (out_w >= 0 && out_w < w && out_h >= 0 && out_h < h);
			if (valid) {
				counter++;
				d += delta[out_index];
			}
		}
	}
	if(counter > 0) prev_delta[index] += d / counter;
}



void forward_local_avgpool_layer_gpu(Darknet::Layer & l, Darknet::NetworkState state)
{
	TAT(TATPARMS);

#ifdef CUDNN_DISABLED
	if (!state.train && l.stride == l.size)
	{
		// hipdnnPoolingBackward
		hipdnnStatus_t maxpool_status;

		float alpha = 1, beta = 0;
		maxpool_status = hipdnnPoolingForward(
			cudnn_handle(),
			l.poolingDesc,
			&alpha,
			l.srcTensorDesc,
			state.input,
			&beta,
			l.dstTensorDesc,
			l.output_gpu);

		//maxpool_status = hipdnnDestroyPoolingDescriptor(poolingDesc);
		//hipdnnDestroyTensorDescriptor(l.srcTensorDesc);
		//hipdnnDestroyTensorDescriptor(l.dstTensorDesc);
	}
	else
#endif
	{
		int h = l.out_h;
		int w = l.out_w;
		int c = l.out_c;

		size_t n = h*w*c*l.batch;

		forward_local_avgpool_layer_kernel <<<cuda_gridsize(n), BLOCK, 0, get_cuda_stream() >>> (n, l.h, l.w, l.c, l.stride_x, l.stride_y, l.size, l.pad, state.input, l.output_gpu);
		CHECK_CUDA(hipPeekAtLastError());
	}
}

void backward_local_avgpool_layer_gpu(Darknet::Layer & l, Darknet::NetworkState state)
{
	TAT(TATPARMS);

	size_t n = l.h * l.w * l.c * l.batch;

	backward_local_avgpool_layer_kernel <<<cuda_gridsize(n), BLOCK, 0, get_cuda_stream() >>>(n, l.h, l.w, l.c, l.stride_x, l.stride_y, l.size, l.pad, l.delta_gpu, state.delta);
	CHECK_CUDA(hipPeekAtLastError());
}
